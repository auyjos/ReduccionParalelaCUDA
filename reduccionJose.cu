#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Variables para N y BLOCK_SIZE que se leerán del archivo
int N;          // Número de elementos en el array
int BLOCK_SIZE; // Tamaño del bloque

__global__ void reductionKernel(float *d_input, float *d_output, int n)
{
    extern __shared__ float sharedData[]; // Array compartido en la GPU
    int tid = threadIdx.x;                // ID del hilo
    int index = blockIdx.x * blockDim.x + tid;

    // Cargar datos en el array compartido
    sharedData[tid] = (index < n) ? d_input[index] : 0.0f;
    __syncthreads(); // Sincronizar hilos en el bloque

    // Reducción en el bloque
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads(); // Sincronizar hilos en el bloque solo si hay más de un hilo
    }

    // Escribir el resultado del bloque en la salida
    if (tid == 0)
    {
        d_output[blockIdx.x] = sharedData[0];
    }
}

// Función para leer los parámetros desde el archivo de texto
int leerParametros(const char *nombreArchivo)
{
    FILE *archivo = fopen(nombreArchivo, "r");
    if (archivo == NULL)
    {
        printf("No se pudo abrir el archivo %s\n", nombreArchivo);
        return -1;
    }

    // Leer los valores de N y BLOCK_SIZE desde el archivo
    if (fscanf(archivo, "%d", &N) != 1 || fscanf(archivo, "%d", &BLOCK_SIZE) != 1)
    {
        printf("Error al leer los parámetros del archivo.\n");
        fclose(archivo);
        return -1;
    }

    fclose(archivo);
    return 0;
}

int main()
{
    // Leer los parámetros desde el archivo de texto
    if (leerParametros("parameters.txt") != 0)
    {
        return -1;
    }

    // Mostrar el tamaño del array y el número de bloques
    printf("Tamano del array N: %d\n", N);
    printf("Tamano del bloque BLOCK_SIZE: %d\n", BLOCK_SIZE);

    // Inicializar datos
    float *h_input = (float *)malloc(sizeof(float) * N);
    srand(time(NULL)); // Semilla para números aleatorios

    for (int i = 0; i < N; i++)
    {
        h_input[i] = (float)(rand() % 100); // Asignar valores aleatorios entre 0 y 99
    }

    float *d_input, *d_output;
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;            // Número de bloques
    float *h_output = (float *)malloc(sizeof(float) * numBlocks); // Buffer de salida

    // Asignar memoria en la GPU
    hipMalloc(&d_input, sizeof(float) * N);
    hipMalloc(&d_output, sizeof(float) * numBlocks);

    // Copiar datos a la GPU
    hipMemcpy(d_input, h_input, sizeof(float) * N, hipMemcpyHostToDevice);

    // Crear eventos para temporización
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Iniciar temporización
    hipEventRecord(start, 0);

    // Configurar el tamaño de los bloques y la grilla
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(numBlocks);

    // Lanzar el kernel con memoria compartida dinámica
    reductionKernel<<<gridSize, blockSize, BLOCK_SIZE * sizeof(float)>>>(d_input, d_output, N);
    hipDeviceSynchronize(); // Esperar a que el kernel termine de ejecutarse

    // Detener temporización
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calcular el tiempo transcurrido
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copiar el resultado de vuelta a la CPU
    hipMemcpy(h_output, d_output, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);

    // Sumar los resultados de los bloques
    float totalSum = 0.0f;
    for (int i = 0; i < numBlocks; i++)
    {
        totalSum += h_output[i];
    }

    // Calcular la suma secuencial para validar la implementación
    float sequentialSum = 0.0f;
    for (int i = 0; i < N; i++)
    {
        sequentialSum += h_input[i];
    }

    // Mostrar los resultados
    printf("Suma total (CUDA): %f\n", totalSum);
    printf("Suma total (secuencial): %f\n", sequentialSum);
    printf("Tiempo de ejecucion (CUDA): %f ms\n", milliseconds);

    // Validar el resultado
    if (fabs(totalSum - sequentialSum) < 1e-5) // Comparar con tolerancia
    {
        printf("La implementacion es valida.\n");
    }
    else
    {
        printf("La implementacion es invalida.\n");
    }

    // Liberar memoria
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}
