#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 100000        // Número de elementos en el array
#define BLOCK_SIZE 1024 // Tamaño del bloque

__global__ void reductionKernel(float *d_input, float *d_output, int n)
{
    __shared__ float sharedData[BLOCK_SIZE]; // Array compartido en la GPU
    int tid = threadIdx.x;                   // ID del hilo
    int index = blockIdx.x * blockDim.x + tid;

    // Cargar datos en el array compartido
    sharedData[tid] = (index < n) ? d_input[index] : 0.0f;
    __syncthreads(); // Sincronizar hilos en el bloque

    // Reducción en el bloque
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads(); // Sincronizar hilos en el bloque solo si hay más de un hilo
    }

    // Escribir el resultado del bloque en la salida
    if (tid == 0)
    {
        d_output[blockIdx.x] = sharedData[0];
    }
}

int main()
{
    // Inicializar datos
    float *h_input = (float *)malloc(sizeof(float) * N);
    srand(time(NULL)); // Semilla para números aleatorios

    for (int i = 0; i < N; i++)
    {
        h_input[i] = (float)(rand() % 100); // Asignar valores aleatorios entre 0 y 99
    }

    float *d_input, *d_output;
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;            // Número de bloques
    float *h_output = (float *)malloc(sizeof(float) * numBlocks); // Buffer de salida

    // Asignar memoria en la GPU
    hipMalloc(&d_input, sizeof(float) * N);
    hipMalloc(&d_output, sizeof(float) * numBlocks);

    // Copiar datos a la GPU
    hipMemcpy(d_input, h_input, sizeof(float) * N, hipMemcpyHostToDevice);

    // Crear eventos para temporización
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Iniciar temporización
    hipEventRecord(start, 0);

    // Configurar el tamaño de los bloques y la grilla
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(numBlocks);

    // Lanzar el kernel
    reductionKernel<<<gridSize, blockSize>>>(d_input, d_output, N);
    hipDeviceSynchronize(); // Esperar a que el kernel termine de ejecutarse

    // Detener temporización
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calcular el tiempo transcurrido
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copiar el resultado de vuelta a la CPU
    hipMemcpy(h_output, d_output, sizeof(float) * numBlocks, hipMemcpyDeviceToHost);

    // Sumar los resultados de los bloques
    float totalSum = 0.0f;
    for (int i = 0; i < numBlocks; i++)
    {
        totalSum += h_output[i];
    }

    // Calcular la suma secuencial para validar la implementación
    float sequentialSum = 0.0f;
    for (int i = 0; i < N; i++)
    {
        sequentialSum += h_input[i];
    }

    // Mostrar los resultados
    printf("Suma total (CUDA): %f\n", totalSum);
    printf("Suma total (secuencial): %f\n", sequentialSum);
    printf("Tiempo de ejecucion (CUDA): %f ms\n", milliseconds);

    // Validar el resultado
    if (fabs(totalSum - sequentialSum) < 1e-5) // Comparar con tolerancia
    {
        printf("La implementacion es valida.\n");
    }
    else
    {
        printf("La implementacion es invalida.\n");
    }

    // Liberar memoria
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}
